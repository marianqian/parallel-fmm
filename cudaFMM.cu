#include <hip/hip_runtime.h>

#include <driver_functions.h>

// from local_expansion.hpp 

//kernel -- all threads run same code in a block/warp
// __shared__ can only be accessed by threads in same block 
__global__ void fmm_kernel(int num_nodes, int num_interact){ 
    int index = blockIdx.x * blockDim.x + threadIdx.x; 
    int node = blockIdx.x; 
    int interaction_node = blockDim.x; 
    int coefficient = threadIdx.x; 



    __syncthreads(); 






}



void FMMCuda(int num_nodes, int num_interact){
    const int threadsPerBlock = 10; 
    const int blocks = ((num_nodes * num_interact) + threadsPerBlock - 1) / threadsPerBlock; 
    // N = num interaction node list * num  nodes 

    fmm_kernel<<<blocks, threadsPerBlock>>>(num_nodes, num_interact); 


}

/*
#pragma omp parallel for
        for(std::size_t i = 0; i < n_nodes_at_depth; i++) {

            FmmNode& current_node = nodes[offset + i]; 

            this->localToLocal(current_node);
            this->multipoleToLocal(current_node);
            template<std::size_t d, bool field_type>
            template<typename FmmNode>
            void AbstractFmmTree<d, field_type>::multipoleToLocal(FmmNode& node) {
            
                std::vector<const ME*> incoming; //vector of incoming multipole expansions
                for(FmmNode* interaction_partner : node.interaction_list) {
                    incoming.push_back(&interaction_partner->multipole_expansion);  
                }

                if(incoming.size() > 0) {
                    node.local_expansion += LE(node.center, incoming); 
                    LocalExpansion<2>::LocalExpansion(const Vector_<2>& center, 
                        std::vector<const ME*> expansions): Super(center, expansions.at(0)->order) {
                    
                    for(const ME* me : expansions) { 
                        *this += LocalExpansion(center, *me); 
                        LocalExpansion<2>::LocalExpansion(const Vector_<2>& center, 
                            const ME& incoming): Super(center, incoming.order) {
                    
                        assert(incoming.order > 0); 
                    
                        this->coefficients = multipoleToLocal(incoming); 
                                                
                        std::vector<Complex> LocalExpansion<2>::multipoleToLocal(
                            const ME& incoming) const {

                        Complex z0 = incoming.center - this->center; // ME center rel. to this->center
                        std::vector<Complex> coefficients(this->order + 1); 

                        const tables::BinomialTable& binomial_table = Super::binomial_table; 
                        const tables::PowTable<Complex> z0_inv_pow_table(1./z0, this->order);   


                        coefficients[0] = incoming(0) * std::log(-z0); 
                        for(int k = 1; k <= this->order; ++k) {
                            double sign = k % 2 == 0 ? 1 : -1;  
                            coefficients[0] += sign * incoming(k) * z0_inv_pow_table(k); // [(4.18), 1] 
                        }

                        // Compute b_l for 1 <= l <= order
                        for(int l = 1; l <= this->order; ++l) {

                            Complex b_l = -incoming(0)/(double)l; 

                            for(int k = 1; k < this->order; ++k) { 
                                double sign = k % 2 == 0 ? 1 : -1;  
                                b_l += sign * incoming(k) * z0_inv_pow_table(k)   
                                    * binomial_table(l+k-1, k-1); // [(4.19), 1]
                            }

                            b_l *= z0_inv_pow_table(l); // [(4.19), 1]
                            coefficients[l] = b_l; 
                        }

                        return coefficients;
                        }
                    }
                    }
                }
                }
            }

        }

*/